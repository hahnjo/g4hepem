#include "hip/hip_runtime.h"

#include "Declaration.hh"

#include "G4HepEmData.hh"
#include "G4HepEmElectronData.hh"

#include <hip/hip_runtime.h>
#include "G4HepEmCuUtils.hh"

// Pull in implementation
#include "G4HepEmElectronInteractionBrem.icc"

//
// Note: both specialisations (needed to be called from the host) are done in 
//  this .cu file below in the TestResMacXSecDataOnDevice function.
template <bool TisSBModel>
__global__
void TestElemSelectorDataBremKernel ( const struct G4HepEmElectronDataOnDevice* theElectronData_d, 
     int* tsInImc_d, double* tsInEkin_d, double* tsInLogEkin_d, double* tsInRngVals_d, 
     int* tsOutRes_d, int numTestCases ) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < numTestCases) {
      // the matrial-cut index
      int imc = tsInImc_d[tid];
      // get start index of the data for this material-cut: 
      // NOTE: start index = -1 in case of single element material, i.e. no selector
      int i0 = TisSBModel ? theElectronData_d->fElemSelectorBremSBDataStart[imc] : theElectronData_d->fElemSelectorBremRBDataStart[imc];
      // NOTE: one should try to avoid to call this kernel for materials with single element !!!
      if ( i0 < 0 ) {
        tsOutRes_d[tid] = 0;
      } else {
        int     numElem = theElectronData_d->fElemSelectorNumElements[imc];
        int     numData;
        double  logE0;
        double  invLD;
        double* xdata;
        if (TisSBModel) {
          numData = theElectronData_d->fElemSelectorNumBremSBData[imc];
          logE0   = theElectronData_d->fElemSelectorBremSBAuxData[2*imc];
          invLD   = theElectronData_d->fElemSelectorBremSBAuxData[2*imc+1];
          xdata   = &(theElectronData_d->fElemSelectorBremSBData[i0]);
        } else {
          numData = theElectronData_d->fElemSelectorNumBremRBData[imc];
          logE0   = theElectronData_d->fElemSelectorBremRBAuxData[2*imc];
          invLD   = theElectronData_d->fElemSelectorBremRBAuxData[2*imc+1];
          xdata   = &(theElectronData_d->fElemSelectorBremRBData[i0]);
        }
        // make sure that $x \in  [x[0],x[ndata-1]]$
        double xv      = max( xdata[0], min( xdata[ numElem * ( numData - 1 ) ], tsInEkin_d[ tid ] ) );
        // compute the lowerindex of the x bin (idx \in [0,N-2] will be guaranted)
        int idxEkin    = __double2int_rz( max( 0.0, min( (tsInLogEkin_d[tid]  -logE0) * invLD, numData - 2.0 ) ) );
        // the real index position is idxEkin x numElem
        int indx0      = idxEkin * numElem;
        int indx1      = indx0 + numElem;
        // linear interpolation
        double x1      = xdata[ indx0++ ];
        double x2      = xdata[ indx1++ ];
        double dl      = x2-x1;
        double  b      = max( 0., min( 1., (xv - x1) / dl ) );
        int  theElemIndex = 0;
        // discrete probabilities, for selecting a given element, are from element index of 0 till #elements-2
        // NOTE: non-deterministic while loop can be turned to deterministic sampling tables for the underlying 
        //       discrete distributions (using Alias table) and combining them with statistical interpolation.
        while ( theElemIndex < numElem-1 && tsInRngVals_d[tid] > xdata[indx0+theElemIndex]+b*(xdata[indx1+theElemIndex]-xdata[indx0+theElemIndex])) { ++theElemIndex; }
        tsOutRes_d[tid] = theElemIndex;   
      }
  }
}  

template <bool TisSBModel>
__global__
void TestElemSelector ( const struct G4HepEmElectronData* theElectronData_d,
     const struct G4HepEmMatCutData* theMatCutData_d, const struct G4HepEmMaterialData* theMaterialData_d,
     int* tsInImc_d, double* tsInEkin_d, double* tsInLogEkin_d, double* tsInRngVals_d, 
     int* tsOutRes_d, int numTestCases ) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numTestCases; i += blockDim.x * gridDim.x) {
    // get number of elements this material (from the currecnt material-cuts)
    // is composed of
    const int imc = tsInImc_d[i];
    const int indxMaterial = theMatCutData_d->fMatCutData[imc].fHepEmMatIndex;
    const struct G4HepEmMatData& theMatData = theMaterialData_d->fMaterialData[indxMaterial];
    const int numOfElement = theMatData.fNumOfElement;
    int targetElemIndx = 0;
    if (numOfElement > 1) {
      targetElemIndx = SelectTargetAtomBrem( theElectronData_d, imc, tsInEkin_d[i], tsInLogEkin_d[i], tsInRngVals_d[i], TisSBModel);
    }
    tsOutRes_d[i] = targetElemIndx;
  }
}

void TestElemSelectorDataOnDevice ( const struct G4HepEmData* hepEmData, int* tsInImc_h, 
     double* tsInEkin_h, double* tsInLogEkin_h, double* tsInRngVals_h, 
     int* tsOutRes_h, int numTestCases, int indxModel, bool iselectron, bool hostLayout ) {
  //                                   
  // --- Allocate device side memory for the input/output data and copy all input
  //     data from host to device
  int*        tsInImc_d = nullptr;
  double*    tsInEkin_d = nullptr;
  double* tsInLogEkin_d = nullptr;
  double* tsInRngVals_d = nullptr;
  int*       tsOutRes_d = nullptr;
  //
  gpuErrchk ( hipMalloc ( &tsInImc_d,     sizeof( int )    * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsInEkin_d,    sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsInLogEkin_d, sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsInRngVals_d, sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsOutRes_d,    sizeof( int ) * numTestCases ) );
  //
  // --- Copy the input data from host to device (test material-cut index, ekin and log-ekin arrays)
  gpuErrchk ( hipMemcpy ( tsInImc_d,     tsInImc_h,     sizeof( int )    * numTestCases, hipMemcpyHostToDevice) );
  gpuErrchk ( hipMemcpy ( tsInEkin_d,    tsInEkin_h,    sizeof( double ) * numTestCases, hipMemcpyHostToDevice) );
  gpuErrchk ( hipMemcpy ( tsInLogEkin_d, tsInLogEkin_h, sizeof( double ) * numTestCases, hipMemcpyHostToDevice) );
  gpuErrchk ( hipMemcpy ( tsInRngVals_d, tsInRngVals_h, sizeof( double ) * numTestCases, hipMemcpyHostToDevice) );
  //
  // --- Launch the kernels
  int numThreads = 512;
  int numBlocks  = std::ceil( float(numTestCases)/numThreads );
  if (hostLayout) {
    const struct G4HepEmElectronData* theElectronData = iselectron ? hepEmData->fTheElectronData_gpu_hl : hepEmData->fThePositronData_gpu_hl;
    const struct G4HepEmMatCutData* theMatCutData = hepEmData->fTheMatCutData_gpu;
    const struct G4HepEmMaterialData* theMaterialData = hepEmData->fTheMaterialData_gpu;
    switch (indxModel) {
      case 0: // not used
        break;
      
      case 1:
        
        TestElemSelector <  true > <<< numBlocks, numThreads >>> (theElectronData, theMatCutData, theMaterialData, tsInImc_d, tsInEkin_d, tsInLogEkin_d, tsInRngVals_d, tsOutRes_d, numTestCases );
        break;
        
      case 2:  
        TestElemSelector < false > <<< numBlocks, numThreads >>> (theElectronData, theMatCutData, theMaterialData, tsInImc_d, tsInEkin_d, tsInLogEkin_d, tsInRngVals_d, tsOutRes_d, numTestCases );
        break;
    }
  } else {
    const struct G4HepEmElectronDataOnDevice* theElectronData = iselectron ? hepEmData->fTheElectronData_gpu : hepEmData->fThePositronData_gpu;
    switch (indxModel) {
      case 0: // not used
        break;
      
      case 1:
        
        TestElemSelectorDataBremKernel <  true > <<< numBlocks, numThreads >>> (theElectronData, tsInImc_d, tsInEkin_d, tsInLogEkin_d, tsInRngVals_d, tsOutRes_d, numTestCases );
        break;
        
      case 2:  
        TestElemSelectorDataBremKernel < false > <<< numBlocks, numThreads >>> (theElectronData, tsInImc_d, tsInEkin_d, tsInLogEkin_d, tsInRngVals_d, tsOutRes_d, numTestCases );
        break;
    }
  }
  //  
  // --- Synchronize to make sure that completed on the device
  hipDeviceSynchronize();
  //
  // --- Copy the results from the device to the host
  gpuErrchk ( hipMemcpy ( tsOutRes_h, tsOutRes_d, sizeof( int ) * numTestCases, hipMemcpyDeviceToHost ) );
  //
  // --- Free all dynamically allocated (device side) memory
  hipFree ( tsInImc_d     );
  hipFree ( tsInEkin_d    );
  hipFree ( tsInLogEkin_d );
  hipFree ( tsInRngVals_d );
  hipFree ( tsOutRes_d    );
}



